#include "hip/hip_runtime.h"
#include "cv.h"
#include "highgui.h"
#include <stdio.h>
#include <math.h>
#include <iostream>
#include <string>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <opencv2/gpu/gpu.hpp>

using namespace std;
using namespace cv;
using namespace cv::gpu;

int drange = 2;
int width = 10;
int height = 10;
cv::Size s = cv::Size(height,width);
int N = width*height;

static __global__ void mykernel(cv::gpu::PtrStepSz<float> src, cv::gpu::PtrStep<float> dst){
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if( x>= src.cols || y >= src.rows) return;

	dst(y,x) = src(y,x) + 2.0;
//	float a = src(y,x) + 2.0;
//	dst.ptr(y)[x] = a;
	return;
}

int d_caller (const cv::gpu::PtrStepSz<float>& src, const cv::gpu::PtrStepSz<float>& dst) {

	unsigned int block_dimX = (unsigned int)std::ceil(src.cols/(float)8);
	unsigned int block_dimY = (unsigned int)std::ceil(src.rows/(float)8);
	dim3 blocks(block_dimX,block_dimY);
	dim3 threads(8,8);
	mykernel<<<blocks, threads>>>(src, dst);
	hipDeviceSynchronize();
//	}
		//check for error
		hipError_t err;
		err = hipGetLastError();
		if(err != hipSuccess){
			std::cout << "mykernel: CUDA error with kernel: " << hipGetErrorString(err) << std::endl;
			return 1;
		}
		return 0;
}

int main(int argc, char **argv)
{
	//hipError_t result;
	cv::Mat* h_in = new cv::Mat[drange];
	cv::Mat* h_out = new cv::Mat[drange];
	cv::gpu::GpuMat* h_gpin = new cv::gpu::GpuMat[drange];
	cv::gpu::GpuMat* d_gpout = new cv::gpu::GpuMat[drange];

	for(int i =0 ; i<drange ; i++){
		h_in[i] = cv::Mat(height,width, CV_32F, cv::Scalar::all(1.0));
		h_out[i] = cv::Mat(height,width, CV_32F);
		h_gpin[i] = cv::gpu::GpuMat(height, width, CV_32F);
		createContinuous(height, width, CV_32F, d_gpout[i]);
		h_gpin[i].upload(h_in[i]);
	}

/*	result = hipMalloc((void**)&d_gpin,drange*width*height*sizeof(float));
	result = hipMalloc((void**)&d_gpout,drange*width*height*sizeof(float));
	if (result != hipSuccess) {
		fprintf(stderr, "Failed to allocate array of GpuMat on device - %s\n", hipGetErrorString(result));
		return 1;
	}

	result = hipMemcpy(d_gpin, h_gpin, drange*sizeof(cv::Mat), hipMemcpyHostToDevice);

	if (result != hipSuccess) {
		fprintf(stderr, "Failed to copy array of gpumats from host to device- %s\n", hipGetErrorString(result));
		return 1;
	}
*/

	for(int i=0; i<drange; i++){
		if (d_caller(h_gpin[i] , d_gpout[i])!=0) {
			std::cout << "kernel call unsuccessful! " << std::endl;
			return -1;
		}
		d_gpout[i].download(h_out[i]);
		std::cout << "hout["<< i << "]= " << h_out[i] << std::endl;
	}
	//result = hipMemcpy(h_gpout, d_gpout, drange*sizeof(cv::Mat), hipMemcpyDeviceToHost);

	return 0;
}


