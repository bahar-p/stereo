#include "cv.h"
#include "highgui.h"
#include <stdio.h>
#include <math.h>
#include <iostream>
#include <string>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <opencv2/gpu/gpu.hpp>


using namespace std;
using namespace cv;
using namespace cv::gpu;

int drange = 2;
int width = 10;
int height = 10;
cv::Size s = cv::Size(height,width);

int main(int argc, char **argv)
{
	hipError_t result;
	//size_t mysize = width * height * drange * sizeof(float);
	//cv::Size size = cv::Size(height, width);
	cv::Mat* h_in = new cv::Mat[drange];
	cv::gpu::GpuMat* h_gpin = new cv::gpu::GpuMat[drange];
	cv::gpu::GpuMat* d_gpin = new cv::gpu::GpuMat[drange];
	cv::gpu::GpuMat* d_out = new cv::gpu::GpuMat[drange];

//	h_in = (struct mtype*) malloc(drange*sizeof(struct mtype)); 
	for(int i =0 ; i<drange ; i++){
		h_in[i] = cv::Mat(height,width, CV_32F, cv::Scalar::all(1.0));
		h_gpin[i] = cv::gpu::GpuMat(height, width, CV_32F);
		d_gpin[i] = cv::gpu::GpuMat(height, width, CV_32F);
		d_out[i] = cv::gpu::GpuMat(height, width, CV_32F);
		//cv::gpu::GpuMat plane;
		//plane.upload(h_in[i]);
		h_gpin[i].upload(h_in[i]);
	}

	result = hipMalloc((void**)&d_gpin,drange*width*height*sizeof(float));

	if (result != hipSuccess) {
		fprintf(stderr, "Failed to allocate array of GpuMat on device - %s\n", hipGetErrorString(result));
		return 1;
	}
	
	result = hipMemcpy(d_gpin, h_gpin, drange*sizeof(cv::Mat), hipMemcpyHostToDevice);

	if (result != hipSuccess) {
		fprintf(stderr, "Failed to copy array of gpumats from host to device- %s\n", hipGetErrorString(result));
		return 1;
	}


	result = hipMemcpy(d_out, d_gpin, drange*sizeof(cv::Mat), hipMemcpyDeviceToHost);
	cv::Mat h_out;
	d_out[0].download(h_out);
	std::cout << "test: " << h_out << std::endl;

/*
	//surface<void, cudaSurfaceType3D> surfRef;
	result = cudaBindSurfaceToArray(surfRef, arrayPointer);
	if (result != hipSuccess) {
		fprintf(stderr, "hipBindTextureToArray() - failed to bind texture to array - %s\n", hipGetErrorString(result));
		return 2;
	}

	const dim3 dimBlock(6,6,6);
	const dim3 dimGrid((width + dimBlock.x-1)/ dimBlock.x, (height + dimBlock.y-1)/dimBlock.y, (depth + dimBlock.z-1 )/dimBlock.z);

	costAD<<<dimGrid,dimBlock>>>(d_left, d_right,surfRef, minDisp, maxDisp, subRW, subRH);

	hipMemcpy3DParms params = {0};
	memset(&params, 0, sizeof(params));
	params.dstPtr.pitch = sizeof(float) * width;
	params.dstPtr.ptr = h_out;
	params.dstPtr.xsize = width;
	params.dstPtr.ysize = height;
	params.srcPos.x = 0;
	params.srcPos.y = 0;
	params.srcPos.z = 0;
	params.srcArray = arrayPointer;
	params.dstPos.x = 0;
	params.dstPos.y = 0;
	params.dstPos.z = 0;
	params.extent.width = width;
	params.extent.depth = depth;
	params.extent.height = height;
	params.kind = hipMemcpyDeviceToHost;
	result = hipMemcpy3D(&params);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy3D - failed to copy from array to host buffer for verification - %s\n", hipGetErrorString(result));
		return 1;
	}

	hipDeviceSynchronize();
	for (int k = 0; k <depth; k++) {
		for (int j = 0; j < height; j++) {
			for (int i = 0; i < width; i++) {
				printf(" gpuh_DSIResulft: %f \n", h_out[i + j * width+ k*width*height]);
				h_DSI.at<double>(i,j,k) = (double) h_out[i + j * width + k * width * height];
			}
		}
	}
	hipFreeArray(arrayPointer);*/
	return 0;
}


