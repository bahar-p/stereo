#include "hip/hip_runtime.h"
#include "cv.h"
#include "highgui.h"
#include <stdio.h>
#include <stdlib.h>
#include <libgen.h>
#include <math.h>
#include <iostream>
#include <string>
#include "image.h"

using namespace std;
using namespace cv;
using namespace cv::gpu;

image *img;

int main(int argc, char **argv)
{	
	double minv, maxv;
   	int minDisp=0, maxDisp;
	int LR = 0;
	if(argc < 6 ) {
		cout << "Usage: ./main leftImg rightImg maxDisp focal_l baseline ?LRCheck? ?mask?" << endl;
		return -1;
	}
	Mat mask;
	//Read input images into Matrices
	Mat image_left = imread(argv[1], -1);
   	Mat image_right = imread(argv[2], -1);
	char* fullpath = argv[1];
	char* bname = basename(fullpath);
	//char* x = strtok(bname, ".");
	const std::string fname(reinterpret_cast<char*>(bname));
	//cout << "filename: " << x << " sName: " << sName  << endl;
	maxDisp = atoi(argv[3]);
	float focal = atof(argv[4]);
	float baseline = atof(argv[5]);
	if(argc>6) LR = atoi(argv[6]);
	if(argc>7) mask = imread(argv[7],0);
	Size s = image_left.size();
	img = new image(image_left,image_right, minDisp, maxDisp);
	cv::Mat dispR8;
	int drange = maxDisp-minDisp+1;
	clock_t tStart = clock();
	cv::Mat* h_DSI = new cv::Mat[drange];
	cv::gpu::GpuMat* d_DSI = new cv::gpu::GpuMat[drange];
	cv::gpu::GpuMat imgL;
	imgL.upload(image_left);
	cv::gpu::GpuMat imgR;
	imgR.upload(image_right);

	for(int d=0;d<drange;d++){
		d_DSI[d] = cv::gpu::GpuMat(img_left.size(), CV_64F, Scalar::all(0));
		if(img->costAD_caller(imgL, imgR,d_DSI[d],d)!=0){
			std::cout<< "kernel call unsuccessful! " << std::endl;
			return -1;
		}
		d_DSI[d].download(h_DSI[d]);
		std::cout << "h_DSI[" << d << "]=" << h_DSI[d](Rect(10,10,5,5)) << std::endl;
	}
	img -> costCensus(7,9,1);
	img-> costCensus(7,9,0);
	img->c_census(7,9);
	img->initCost(h_DSI, 10,30);
	img->line_segment(20.,6.,34.,17.);
	img->aggregateCost(h_DSI);
	for(int i=0;i< maxDisp-minDisp+1 ; i++)
		h_DSI[i].release();
	Mat dispL= cv::Mat(s.height, s.width, CV_32FC1,cv::Scalar::all(0));
	Mat costL= cv::Mat(s.height, s.width, CV_32FC1,cv::Scalar::all(0));
	cv::Mat* fcost = img->scanline(1.0,3.0,15, dispL, costL);
	
/*	if(LR){
		bool Rdisp= true;
		img->reset();
		h_DSI = img->costAD(Rdisp);
		img->c_census(7,9,Rdisp);
		img->initCost(DSI, 10,30);
		img->line_segment(20.,6.,34.,17.,Rdisp);
		img->aggregateCost(DSI);
		Mat dispR=cv::Mat(s.height, s.width, CV_32FC1,cv::Scalar::all(0));
		Mat costR=cv::Mat(s.height, s.width, CV_32FC1,cv::Scalar::all(0));
		img->scanline(1.0,3.0,15, dispR, costR,Rdisp);
		cv::minMaxLoc(dispR, &minv,&maxv);
		//dispR8 = Mat(dispR.size().height, dispR.size().width, CV_8UC1, Scalar::all(0));
		//dispR.convertTo( dispR8, CV_8UC1,255.0/maxDisp);
		//std::cout << "Execution time:  " << double( clock() - tStart) / (double)CLOCKS_PER_SEC<< " seconds." << std::endl;
		/* Refinement */	
	/*	cv::Mat pixflags(dispL.rows, dispL.cols,CV_32S, Scalar::all(0));
		img->findOutliers(dispL, dispR,pixflags,focal, baseline);
		img->regionVoting(dispL, pixflags, 20, 0.4, 5);
		img->findOutliers(dispL, dispR,pixflags,focal, baseline);
		img->interpolate(image_left, dispL, pixflags);
		Mat br;
		img->border(dispL, br);
		img->discAdjust(dispL, fcost, br);
		imshow( "borders", br );                   	
	}*/
	img->subpxEnhance(fcost,dispL);
	//cerr << "out of subPx" << endl;
	std::cout << "Exec_time: " << double( clock() - tStart) / (double)CLOCKS_PER_SEC<< " seconds." << std::endl;
	double minv1, maxv1;
	cv::minMaxLoc(dispL, &minv1,&maxv1);
	Mat dispL8;
	//cout << "maxv: " << maxv1 << endl;
	//cout << "final disp channels: " << dispL.channels() << " depth: " << dispL.depth() << endl;
	dispL.convertTo( dispL8, CV_8UC1,255.0/maxDisp);
//	imshow( "Img", image_left );                   
	imshow( "DispL", dispL8 );                   	
//	if(LR) imshow( "DispR", dispR8 ); 
	string fpath1 = "/home/bahar/Master/stereo/Ex1/adcensus/mydisp/" + fname;
	imwrite(fpath1 , dispL8);
	if(argc>7) {
		Mat d_masked;
		string fpath2 = "/home/bahar/Master/stereo/Ex1/adcensus/dispmasked/" + fname;
		dispL8.copyTo(d_masked, mask);
		imshow( "DispMasked", d_masked );                   	
		imwrite(fpath2, d_masked);
	}
	waitKey(0);
   //	char c = waitKey(10);
//	if (c == ' ')  return 0;
	return 0;
}

