#include "hip/hip_runtime.h"
#include "cv.h"
#include "highgui.h"
#include <stdio.h>
#include <math.h>
#include <iostream>
#include <string>
#include "imageGpu.h"

using namespace std;
using namespace cv;
using namespace cv::gpu;


imageGpu *img;

int main(int argc, char **argv)
{
	float focal = 3740;				//pixel
	float baseline = 160;			//milimeter	
	
	/*cv::Mat p1= (Mat_<float>(3,1) << 1,1,0);
	cv::Mat q1= (Mat_<float>(3,1) << 2,3,1);
	cv::Mat res1;
	gemm(p1,q1, 1, 0, 0, res1, GEMM_2_T);
	cout<< "p1= " << p1 << '\t' << " q1= " << q1 << '\t' << " res1= " << res1 << endl;*/
	
	//Read input images into Matrices
	Mat image_left = imread(argv[1], CV_LOAD_IMAGE_COLOR);
   	Mat image_right = imread(argv[2], CV_LOAD_IMAGE_COLOR);
   	//Mat image_left = Mat(6,6, CV_8U, Scalar::all(0));
   	//Mat image_right = Mat(6,6, CV_8U, Scalar::all(0));
	gpu::GpuMat d_left, d_right;
	Size s = image_left.size();
   	int minDisp=0, maxDisp=17;
    	img = new imageGpu(image_left,image_right, minDisp, maxDisp);
	int width = s.width;
	int height = s.height;
	int depth = maxDisp-minDisp+1;
	
	int h_sz[] = {height, width, depth};
	cv::Mat h_DSI(3, h_sz,CV_64FC1, cv::Scalar::all(0));

	clock_t tStart = clock();
	img->reset();
	
	img->copyHosttoDevice(image_left, d_left);
	img->copyHosttoDevice(image_right , d_right);

	hipError_t result;
	hipArray *arrayPointer;
	size_t bytes = width * height * depth * sizeof(float);
	float *h_out;
	float *d_out;

	h_out = (float *)malloc(bytes);
	surface<void,cudaSurfaceType3D> surfRef;
	// initial value
	for (int k = 0; k <depth; k++) {
		for (int j = 0; j < height; j++) {
			for (int i = 0; i < width; i++) {
				h_out[i + j * width + k * width * height] = -100;
			}
		}
	}

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipExtent extent;
	extent.width = width;
	extent.depth = depth;
	extent.height = height;
	result = hipMalloc3DArray(&arrayPointer, &channelDesc, extent, hipArraySurfaceLoadStore);
	if (result != hipSuccess) {
		fprintf(stderr, "Texture3D - failed to malloc 3D array - %s\n", hipGetErrorString(result));
		return 1;
	}

	//surface<void, cudaSurfaceType3D> surfRef;
	result = cudaBindSurfaceToArray(surfRef, arrayPointer);
	if (result != hipSuccess) {
		fprintf(stderr, "hipBindTextureToArray() - failed to bind texture to array - %s\n", hipGetErrorString(result));
		return 2;
	}

	const dim3 dimBlock(6,6,6);
	const dim3 dimGrid((width + dimBlock.x-1)/ dimBlock.x, (height + dimBlock.y-1)/dimBlock.y, (depth + dimBlock.z-1 )/dimBlock.z);

	costAD<<<dimGrid,dimBlock>>>(d_left, d_right,surfRef, minDisp, maxDisp, subRW, subRH);

	hipMemcpy3DParms params = {0};
	memset(&params, 0, sizeof(params));
	params.dstPtr.pitch = sizeof(float) * width;
	params.dstPtr.ptr = h_out;
	params.dstPtr.xsize = width;
	params.dstPtr.ysize = height;
	params.srcPos.x = 0;
	params.srcPos.y = 0;
	params.srcPos.z = 0;
	params.srcArray = arrayPointer;
	params.dstPos.x = 0;
	params.dstPos.y = 0;
	params.dstPos.z = 0;
	params.extent.width = width;
	params.extent.depth = depth;
	params.extent.height = height;
	params.kind = hipMemcpyDeviceToHost;
	result = hipMemcpy3D(&params);
	if (result != hipSuccess) {
		fprintf(stderr, "hipMemcpy3D - failed to copy from array to host buffer for verification - %s\n", hipGetErrorString(result));
		return 1;
	}

	hipDeviceSynchronize();
	for (int k = 0; k <depth; k++) {
		for (int j = 0; j < height; j++) {
			for (int i = 0; i < width; i++) {
				printf(" gpuh_DSIResulft: %f \n", h_out[i + j * width+ k*width*height]);
				h_DSI.at<double>(i,j,k) = (double) h_out[i + j * width + k * width * height];
			}
		}
	}


/*
	img->c_census(7,9);
	img->initCost(10,30);
	img->line_segment(20.,6.,34.,17.);
	img->aggregateCost();
	
	Mat dispL=cv::Mat(s.height, s.width, CV_32FC1,cv::Scalar::all(0));
	Mat costL=cv::Mat(s.height, s.width, CV_32FC1,cv::Scalar::all(0));
	Mat fcost = img->scanline(1.0,3.0,15, dispL, costL);
	std::cout << "Execution time:  " << double( clock() - tStart) / (double)CLOCKS_PER_SEC<< " seconds." << std::endl;
	
	double minv1, maxv1;                    
	cv::minMaxLoc(dispL, &minv1,&maxv1);
	//Mat dispL8 = Mat(dispL.size().height, dispL.size().width, CV_8UC1, Scalar::all(0));
	Mat dispL8;
	dispL.convertTo( dispL8, CV_8UC1,255.0/maxDisp);
 
	
    imshow( "DispL", dispL8 );                   	
   	
    waitKey(0);
   	char c = waitKey(10);
    if (c == ' ')  */

	hipFreeArray(arrayPointer);
	return 0;
}


