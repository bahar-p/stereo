#include "hip/hip_runtime.h"
#include "imageGpu.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <opencv2/gpu/gpu.hpp>


using namespace std;
using namespace cv;
using namespace cv::gpu;


//__global__ void costAD(gpu::PtrStepSz<uchar3>& d_left, gpu::PtrStepSz<uchar3>& d_right, surface<void, cudaSurfaceType3D> outSurfObj, int dispMin, int dispMax, int subRW, 
//				int subRH, bool dispR);

imageGpu::imageGpu(Mat image_leftRGB, Mat image_rightRGB, int dMin, int dMax){
	img_leftRGB=image_leftRGB;
	img_rightRGB=image_rightRGB;
	dispMin = dMin;
	dispMax = dMax;
	Size s = img_leftRGB.size();
	
	int sz[] = {s.height, s.width, dispMax-dispMin+1};
	
	//Initialization of different cost space
	
	DSI=cv::Mat(3, sz, mytype,cv::Scalar::all(0));
	init_cost=cv::Mat(3, sz, mytype,cv::Scalar::all(0));
	aggr_cost=cv::Mat(3, sz, mytype, cv::Scalar::all(0));
	final_cost=cv::Mat(3, sz, mytype, cv::Scalar::all(0));
	left_cost=cv::Mat(3, sz, mytype, cv::Scalar::all(0));
	right_cost=cv::Mat(3, sz, mytype, cv::Scalar::all(0));
	up_cost=cv::Mat(3, sz, mytype, cv::Scalar::all(0));
	down_cost=cv::Mat(3, sz, mytype, cv::Scalar::all(0));
	HII=cv::Mat(3, sz , mytype, cv::Scalar::all(0));
	VII=cv::Mat(3, sz, mytype, cv::Scalar::all(0));
	sumH=cv::Mat(3, sz, mytype, cv::Scalar::all(0));
	sumV=cv::Mat(3, sz, mytype, cv::Scalar::all(0));
	census_hamming = (unsigned***)malloc(img_leftRGB.rows*sizeof(unsigned**));
	censusLeft = (uint64_t**) malloc(img_leftRGB.rows*sizeof(uint64_t*));
	censusRight = (uint64_t**) malloc(img_leftRGB.rows*sizeof(uint64_t*));
	
	int d[]={s.height, s.width, 4};
	supReg=cv::Mat(3, d , CV_32SC1, cv::Scalar::all(0));
	
	for (int i = 0 ;  i < img_leftRGB.rows; i++) {
		censusLeft[i] = (uint64_t*) malloc(sizeof(uint64_t) * img_leftRGB.cols);			//Left image Census Cost
		censusRight[i] = (uint64_t*) malloc(sizeof(uint64_t) * img_leftRGB.cols);			//Right image Census Cost
		census_hamming[i] = (unsigned**) malloc(sizeof(unsigned*) * img_leftRGB.cols);		//Hamming distance for census cost
		for (int j = 0; j < img_leftRGB.cols; j++){
			census_hamming[i][j]= (unsigned *)malloc(sizeof(unsigned) * (dispMax-dispMin+1));
		}
	}
	for(int p= 0 ; p<img_leftRGB.rows ; p++){
		for(int q= 0 ; q<img_leftRGB.cols ; q++){
			censusLeft[p][q]=0;
			censusRight[p][q]=0;
			for(int d=0;d<dispMax-dispMin+1;d++){		
				census_hamming[p][q][d]=0;
			}
		}
	}
	
}

void imageGpu::copyHosttoDevice(cv::Mat src, gpu::GpuMat& dst){
	
	dst.upload(src);
}
void imageGpu::copyDevicetoHost(gpu::GpuMat src, cv::Mat& dst){
	src.download(dst);
}

void imageGpu::reset(){
	DSI=cv::Scalar::all(0);
	init_cost=cv::Scalar::all(0);
	aggr_cost=cv::Scalar::all(0);
	final_cost=cv::Scalar::all(0);
	left_cost=cv::Scalar::all(0);
	right_cost=cv::Scalar::all(0);
	up_cost=cv::Scalar::all(0);
	down_cost=cv::Scalar::all(0);
	HII= cv::Scalar::all(0);
	VII=cv::Scalar::all(0);
	sumH= cv::Scalar::all(0);
	sumV=cv::Scalar::all(0);
	supReg=cv::Scalar::all(0);
	for(int p= 0 ; p<img_leftRGB.rows ; p++){
		for(int q= 0 ; q<img_leftRGB.cols ; q++){
			for(int d=0;d<dispMax-dispMin+1;d++){		
				census_hamming[p][q][d]=0;
			}
		}
	}
	
}
Mat imageGpu::get_image(int left){
	if(left==1)
		return img_leftRGB;
	else return img_rightRGB;
}

/* Calculating the average intesity difference for each pixel and its correspondence */
__global__ void costAD(gpu::PtrStepSz<uchar3> d_left, gpu::PtrStepSz<uchar3> d_right,surface<void,cudaSurfaceType3D> surfRef, int dmin, int dmax, int rw, int rh)
{
	
	int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y = blockIdx.y*blockDim.y+threadIdx.y;
	int z = blockIdx.z*blockDim.z+threadIdx.z;

	if(x >= d_left.cols || y >= d_left.rows || z >= dmax-dmin+1) return;
	
//	if(dispR==false){
		if(x-z-dmin > rw-1){
			float data =(float) (fabs((double)d_left(y,x).x - (double)d_right(y,x-z-dmin).x) + fabs((double)d_left(y,x).y - (double)d_right(y,x-z-dmin).y) + 
					(double) fabs(d_left(y,x).z - (double) d_right(y,x-z-dmin).z))/3.0;
			surf3Dwrite(data,surfRef,x*4,y,z,hipBoundaryModeTrap);
			 /* d_DSI.at<double>(p,q,d)= (fabs(d_leftRGB.at<cv::Vec3b>(p,q).val[0] - d_rightRGB.at<cv::Vec3b>(p,q-d-dispMin).val[0]) + 
						(fabs(d_leftRGB.at<cv::Vec3b>(p,q).val[1] - d_rightRGB.at<cv::Vec3b>(p,q-d-dispMin).val[1])) +
						(fabs(d_leftRGB.at<cv::Vec3b>(p,q).val[2] - d_rightRGB.at<cv::Vec3b>(p,q-d-dispMin).val[2]))/3.0; */
		}
//	}
/*	else{
		//Right disparity
		if(x+z+dispMin < d_left.cols-subRW){
			float data = (fabs(d_left(y,x+z+dispMin).x - d_right(y,x).x) + fabs(d_left(y,x+z+dispMin).y - d_right(y,x).y) + fabs(d_left(y,x+z+dispMin).z - d_right(y,x).z))/3.0;
			surf3Dwrite(data,outSurfObj,x*4,y,z, hipBoundaryModeTrap);
		}
	}*/
}

/* Encoding each pixel local structure */
void imageGpu::costCensus(int winX, int winY, int left){
	uint64_t census;
	uint64_t bit=0;
	int shifts;
	cv::Mat left_gray, right_gray;
	cvtColor(img_leftRGB,left_gray,CV_BGR2GRAY);
	cvtColor(img_rightRGB,right_gray,CV_BGR2GRAY);
	for(int x = winX/2; x<img_leftRGB.rows - winX/2; x++){
		for(int y = winY/2; y<img_leftRGB.cols - winY/2 ; y++){
			//printf("x: %d\t , y: %d\t" , x,y);
			census=0;
			shifts=0;
			for(int i=x - winX/2; i <= x + winX/2 ; i++){
				for(int j = y - winY/2; j <= y + winY/2 ; j++){
					if(shifts != winX*winY/2){						//just to exclude the central pixel from the calculation
						census <<= 1;	
						if(left==1){								//left image		
							if(left_gray.at<uchar>(i,j)<left_gray.at<uchar>(x,y))
								bit=1;
							else bit=0;
						} else {
							if(right_gray.at<uchar>(i,j)<right_gray.at<uchar>(x,y))
								bit=1;
							else bit=0;
						}
						census = census+bit;
						//printf("census: %lld\n" , census);
					}
					shifts++;
				}
			}
			if(left==1)
				censusLeft[x][y]= census;
			else censusRight[x][y]= census;
			/*if(x==23 && y==60)
				printf("x: %d\t , y: %d\t , censusL: %s\n" , x,y, itob(censusLeft[x][y]));
			if(x==23 && y==58)
				printf("x: %d\t , y: %d\t , censusR: %s\n" , x,y, itob(censusRight[x][y]));*/
		}
	}
}

char * imageGpu::itob(uint64_t x)
{
	cout << sizeof(uint64_t) << endl;
	static char buff[sizeof(uint64_t) * CHAR_BIT + 1];
	int i;
	for(i=0;i<sizeof(uint64_t) * CHAR_BIT; i++)
	{
		uint64_t unit = 1;
		if(x & (unit << (sizeof(uint64_t) * CHAR_BIT -1- i)))
			buff[i] = '1';
		else
			buff[i] = '0';
	}
	cout << "length: " << i << endl;
	return buff;
}

/* Calculating the hamming distance between each pixel and its correspondence census cost */
void imageGpu::hamdist(uint64_t** censL, uint64_t** censR, int winX, int winY, bool dispR){
	int d,p,q;
	uint64_t val=0;
	unsigned dist=0;
	for(d=0;d<dispMax-dispMin+1;d++){
		for(p= winX/2 ; p<img_leftRGB.rows - winX/2 ; p++){					//Rows = height
			for(q= winY/2 ; q<img_leftRGB.cols - winY/2 ; q++){				//cols = width
				dist=0;
				val=0;
				if(!dispR){
					if(q-d-dispMin>subRW -1){
						val = censL[p][q] ^ censR[p][q-d-dispMin];				//XOR operation
						//printf("censL: %lld\t\t , censR: %lld\t\t", censL[p][q+d+dispMin], censR[p][q]);
						while(val){
							++dist;
							val &= val - 1;
						}
						census_hamming[p][q][d]= dist;
						//printf("hammingDist: %u\t\t", dist);
						/*if(p==23 && q==60 && (q-d-dispMin==58)){
							printf("x: %d\t , q: %d\t , census_hamming: %u\n" , p,q-d-dispMin, census_hamming[p][q][d]);
						}*/
					}
				}
				else{
					if(q+d+dispMin<img_leftRGB.cols-subRW){
						val = censL[p][q+d+dispMin] ^ censR[p][q];				//XOR operation
						//printf("censL: %lld\t\t , censR: %lld\t\t", censL[p][q+d+dispMin], censR[p][q]);
						while(val){
							++dist;
							val &= val - 1;
						}
						census_hamming[p][q][d]= dist;
						//printf("hammingDist: %u\t\t", dist);
					}
				}
			}
		}
		
	}
	
	//std::cout<< "census_hamming(100,200,10): " << census_hamming[100][200][10] << std::endl;
}

/* Calculate census cost */
void imageGpu::c_census(int X, int Y, bool dispR){
	
	hamdist(censusLeft, censusRight, X,Y, dispR);
	
}
/* Calculating the initial cost: Census + AD */
void imageGpu::initCost(double lam_AD, double lam_census){
	std::cout<< "AdCensus algorithm..." <<std::endl;
	int p,q,d;
	for(d=0;d<dispMax-dispMin+1;d++){
		for(p= subRH ; p<img_leftRGB.rows-subRH ; p++){					//Rows = height
			for(q= subRW ; q<img_leftRGB.cols-subRW ; q++){				//cols = width
				init_cost.at<double>(p,q,d)= 2.0 - exp(-DSI.at<double>(p,q,d)/lam_AD) - exp(-(double)census_hamming[p][q][d]/lam_census);
				if(init_cost.at<double>(p,q,d) > 2.0)
					abort();
			}
		}
		
	}
	//std::cout<< "init_cost(100,200,10): " << init_cost.at<double>(100,200,10) << std::endl;
	//printf("DSI: %f\t\t , census: %f\t\t , cost: %f\t\t\n ", DSI[0][0][0], (double)census_hamming[0][0][0],init_cost[0][0][0] );
}

/* Building the cross-based region for each pixel */
void imageGpu::line_segment(double colLim1, double colLim2, double distLim1, double distLim2, bool dispR){
	int p,q,x,y;
	bool arm_found=false;
	for(p= subRH ; p<img_leftRGB.rows-subRH; p++){					//Rows = height
		for(q= subRW ; q<img_leftRGB.cols-subRW ; q++){				//cols = width
			
			if(!dispR){
				arm_found=false;
				if(q!=subRW){
					int dist;
					for(y=q-1; y>=subRW; y--){							//scan left arm - The first arg
						double col_diff1 = colDiffer(img_leftRGB, p,y,p,q);
						double col_diff2 = colDiffer(img_leftRGB, p,y,p,y+1);		//the endpoint and its predecessor
						dist= std::abs(y-q);					
						
						if(!(col_diff1<colLim1 && col_diff2<colLim1) || !(dist<distLim1) || (dist<distLim1 && dist>distLim2 && !(col_diff1<colLim2))){	//if violates any of these rules
							 supReg.at<int>(p,q,0)=dist-1;
							 arm_found=true;
							 //std::cout<< "For the point (" << p << " , "  << q << ") The left arm endpoint is: " << dist << std::endl;
							 break;
						}
					}
					if( supReg.at<int>(p,q,0)==0 && !arm_found){							//if no endpoint found and it's not one of the edges' pixels
						//int dist = std::abs(subRW-q);
						supReg.at<int>(p,q,0)= dist;										//The edge pixel will be the end point.
						//std::cout<< "excep: For the point (" << p << " , "  << q << ") The left arm endpoint is: " << dist << std::endl;
					}
				}
				arm_found=false;
				if(q!=img_leftRGB.cols-subRW-1){
					int dist;
					for(y=q+1; y<img_leftRGB.cols - subRW; y++){							//scan right arm - The second arg
						double col_diff1 = colDiffer(img_leftRGB, p,y,p,q);
						double col_diff2 = colDiffer(img_leftRGB,p,y,p,y-1);						//the end point and its predecessor
						dist= std::abs(y-q);
						if(!(col_diff1<colLim1 && col_diff2<colLim1) || !(dist<distLim1) || (dist<distLim1 && dist>distLim2 && !(col_diff1<colLim2))){
							 supReg.at<int>(p,q,1)=dist-1;
							 arm_found=true;
							// std::cout<< "For the point (" << p << " , "  << q << ") The right arm endpoint is: " << dist << std::endl;
							 break;
						}
					}
				
					if( supReg.at<int>(p,q,1)==0 && !arm_found){	
						//int dist = std::abs(img_leftRGB.cols-1-subRW-q);	
						supReg.at<int>(p,q,1)=dist;										
						//std::cout<< "excep: For the point (" << p << " , "  << q << ") The right arm endpoint is: " << dist << std::endl;
					}
				}
				arm_found=false;
				if(p!=subRH){
					int dist;
					for(x=p-1; x>=subRH; x--){											//scan up arm - The third arg
						double col_diff1 = colDiffer(img_leftRGB,x,q,p,q);
						double col_diff2 = colDiffer(img_leftRGB,x,q,x+1,q);
						dist= std::abs(x-p);
						 
						if(!(col_diff1<colLim1 && col_diff2<colLim1) || !(dist<distLim1) || (dist<distLim1 && dist>distLim2 && !(col_diff1<colLim2))){
							  supReg.at<int>(p,q,2)=dist-1;
							  arm_found = true;
							 //std::cout<< "For the point (" << p << " , "  << q << ") The up arm	endpoint is: " << dist << std::endl;
							 break;
						}
					}
				
					if( supReg.at<int>(p,q,2)==0 && !arm_found){
						//int dist = std::abs(subRH-p);
						supReg.at<int>(p,q,2)= dist;										
						//std::cout<< "excep: For the point (" << p << " , "  << q << ") The up arm endpoint is: " << dist << std::endl;
					}
				}
				arm_found=false;
				if(p!=img_leftRGB.rows-subRH-1){
					int dist;
					for(x=p+1; x<img_leftRGB.rows-subRH; x++){							//scan bottom arm - The fourth arg
						double col_diff1 = colDiffer(img_leftRGB,x,q,p,q);
						double col_diff2 = colDiffer(img_leftRGB,x,q,x-1,q);						//the end point and its predecessor
						dist= std::abs(x-p);
						
						if(!(col_diff1<colLim1 && col_diff2<colLim1) || !(dist<distLim1) || (dist<distLim1 && dist>distLim2 && !(col_diff1<colLim2))){
							 supReg.at<int>(p,q,3)=dist-1;
							 arm_found=true;
							//std::cout<< "For the point (" << p << " , "  << q << ") The bottom arm endpoint is: " << dist << std::endl;
							break;
						}
					}
				if( supReg.at<int>(p,q,3)==0 && !arm_found){
					//int dist = std::abs(img_leftRGB.rows-1-subRH-p);
					supReg.at<int>(p,q,3)= dist;
					//std::cout<< "excep: For the point (" << p << " , "  << q << ") The bottom arm endpoint is: " << dist << std::endl;
				}
			}
			} else {
				arm_found=false;
				if(q!=subRW){
					int dist;
					for(y=q-1; y>=subRW; y--){							//scan left arm - The first arg
						double col_diff1 = colDiffer(img_rightRGB, p,y,p,q);
						double col_diff2 = colDiffer(img_rightRGB, p,y,p,y+1);		//the endpoint and its predecessor
						dist= std::abs(y-q);					
						if(!(col_diff1<colLim1 && col_diff2<colLim1) || !(dist<distLim1) || (dist<distLim1 && dist>distLim2 && !(col_diff1<colLim2))){	//if violates any of these rules
							 supReg.at<int>(p,q,0)=dist-1;
							 arm_found = true;
							 //std::cout<< "For the point (" << p << " , "  << q << ") The left arm endpoint is: " << dist << std::endl;
							 break;
						}
					}
					if( supReg.at<int>(p,q,0)==0 && !arm_found){							//if no endpoint found and it's not one of the edges' pixels
						 //int dist = std::abs(subRW-q);
						 supReg.at<int>(p,q,0)= dist;										//The edge pixel will be the end point.
						//std::cout<< "excep: For the point (" << p << " , "  << q << ") The left arm endpoint is: " << dist << std::endl;
					}
				}
				arm_found=false;
				if(q!=img_leftRGB.cols-subRW-1){
					int dist;
					for(y=q+1; y<img_leftRGB.cols - subRW; y++){							//scan right arm - The second arg
						double col_diff1 = colDiffer(img_rightRGB, p,y,p,q);
						double col_diff2 = colDiffer(img_rightRGB,p,y,p,y-1);						//the end point and its predecessor
						dist= std::abs(y-q);
						
						if(!(col_diff1<colLim1 && col_diff2<colLim1) || !(dist<distLim1) || (dist<distLim1 && dist>distLim2 && !(col_diff1<colLim2))){
							  supReg.at<int>(p,q,1)=dist-1;
							  arm_found=true;
							// std::cout<< "For the point (" << p << " , "  << q << ") The right arm endpoint is: " << dist << std::endl;
							 break;
						}
					}
					if( supReg.at<int>(p,q,1)==0 && !arm_found){	
						//int dist = std::abs(img_leftRGB.cols-1-subRW-q);	
						 supReg.at<int>(p,q,1)=dist;										
						//std::cout<< "excep: For the point (" << p << " , "  << q << ") The right arm endpoint is: " << dist << std::endl;
					}
				}
				arm_found=false;
				if(p!=subRH){
					int dist;
					for(x=p-1; x>=subRH; x--){											//scan up arm - The third arg
						double col_diff1 = colDiffer(img_rightRGB,x,q,p,q);
						double col_diff2 = colDiffer(img_rightRGB,x,q,x+1,q);
						dist= std::abs(x-p);
						
						if(!(col_diff1<colLim1 && col_diff2<colLim1) || !(dist<distLim1) || (dist<distLim1 && dist>distLim2 && !(col_diff1<colLim2))){
							  supReg.at<int>(p,q,2)=dist-1;
							  arm_found=true;
							 //std::cout<< "For the point (" << p << " , "  << q << ") The up arm	endpoint is: " << dist << std::endl;
							 break;
						}
					}
					if( supReg.at<int>(p,q,2)==0 && !arm_found){
						//int dist = std::abs(subRH-p);
						 supReg.at<int>(p,q,2)= dist;										
						//std::cout<< "excep: For the point (" << p << " , "  << q << ") The up arm endpoint is: " << dist << std::endl;
					}
				}
				arm_found=false;
				if(p!=img_leftRGB.rows-subRH-1){
					int dist;
					for(x=p+1; x<img_leftRGB.rows-subRH; x++){							//scan bottom arm - The fourth arg
						double col_diff1 = colDiffer(img_rightRGB,x,q,p,q);
						double col_diff2 = colDiffer(img_rightRGB,x,q,x-1,q);						//the end point and its predecessor
						dist= std::abs(x-p);
						
						if(!(col_diff1<colLim1 && col_diff2<colLim1) || !(dist<distLim1) || (dist<distLim1 && dist>distLim2 && !(col_diff1<colLim2))){
							 supReg.at<int>(p,q,3)=dist-1;
							 arm_found=true;
							//std::cout<< "For the point (" << p << " , "  << q << ") The bottom arm endpoint is: " << dist << std::endl;
							break;
						}
					}
					if( supReg.at<int>(p,q,3)==0 && !arm_found){
						//int dist = std::abs(img_leftRGB.rows-1-subRH-p);
						 supReg.at<int>(p,q,3)= dist;
						//std::cout<< "excep: For the point (" << p << " , "  << q << ") The bottom arm endpoint is: " << dist << std::endl;
					}
				}
			}
		}
	}
}

/* Calculating aggregated cost */
void imageGpu::aggregateCost(){
	int counter=1;
	int iter=4;
	
	while(counter<iter+1){
		switch(counter){
			case 1:
				std::cout<<"1"<<std::endl;
				IImage(init_cost, HII, 'H');
				finalSum(HII, sumH, 'H', counter);
				IImage(sumH, VII, 'V');
				finalSum(VII, sumV,'V', counter);
				//VII = cv::Scalar::all(0);
				//std::cout<< "sumV(100,200,10): " << sumV.at<double>(100,200,10) << std::endl;
			break;
			case 2:
				std::cout<<"2"<<std::endl;
				IImage(sumV, VII, 'V');
				finalSum(VII, sumV,'V', counter);
				//VII = cv::Scalar::all(0);
				IImage(sumV, HII, 'H');
				finalSum(HII, sumH, 'H', counter);
				//HII = cv::Scalar::all(0);
				//std::cout<< "sumH(100,200,10): " << sumH.at<double>(100,200,10) << std::endl;
			break;
			case 3:
				std::cout<<"3"<<std::endl;
				IImage(sumH, HII,'H');
				finalSum(HII, sumH, 'H',counter);
				//HII = cv::Scalar::all(0);
				IImage(sumH, VII, 'V');
				finalSum(VII,sumV,'V',counter);
				//VII = cv::Scalar::all(0);
				//std::cout<< "sumV(100,200,10): " << sumV.at<double>(100,200,10) << std::endl;
			break;
			case 4:
				std::cout<< "4"<<std::endl;
				IImage(sumV, VII, 'V');
				finalSum(VII, sumV,'V',counter);
				//VII = cv::Scalar::all(0);
				IImage(sumV, HII, 'H');
				finalSum(HII, sumH, 'H',counter);
				//HII = cv::Scalar::all(0);
				//std::cout<< "sumH(100,200,10): " << sumH.at<double>(100,200,10) << std::endl;
			break;
			default:
				std::cout<< "iteration out of range" <<std::endl;
			break;
		}
		counter++;
	}
	//aggr_cost = cv::Scalar::all(0);
	finalSum(sumH, aggr_cost, 'C',  counter-1);
	//std::cout<< "aggr_cost(100,200,10): " << aggr_cost.at<double>(100,200,10) << std::endl;
		
}
/* Calculating Integral Image */
void imageGpu::IImage(cv::Mat in, cv::Mat out, char dir){
	double max=0.0;
	switch (dir){
		case 'H':
			for(int d=0; d<dispMax-dispMin+1; d++){
				for(int p=subRH ; p<img_leftRGB.rows-subRH ; p++){					
					for(int q= subRW ; q<img_leftRGB.cols-subRW ; q++){
						out.at<double>(p,q,d) =in.at<double>(p,q,d) + out.at<double>(p,q-1,d);
					}
				}
			}
			max = findMax(out);
			std::cout << "Imax: " << max << " dir:H" << std::endl;
		break;
		case 'V':
			for(int d=0; d<dispMax-dispMin+1; d++){
				for(int q= subRW ; q<img_leftRGB.cols-subRW ; q++){
					for(int p=subRH ; p<img_leftRGB.rows-subRH ; p++){	
						out.at<double>(p,q,d)= in.at<double>(p,q,d) + out.at<double>(p-1,q,d);
						//printf("IImage: VII[%d][%d][%d]= %Lf \t\n", p, q, d, out[p][q][d]);
					}
				}
			}
			max = findMax(out);
			std::cout << "Imax: " << max << " dir: V" << std::endl;
		break;
		default:
			std::cout<< "none of the directions!"<< std::endl;
		break;
	}
}

/* Calculating final cost at each stage based on calculated integral image and the local support region for each pixel */
void imageGpu::finalSum(cv::Mat in, cv::Mat out, char dir, int count){
switch (dir){
		case 'H':
			for(int d=0; d<dispMax-dispMin+1; d++){
				for(int p=subRH ; p<img_leftRGB.rows-subRH ; p++){					
					for(int q= subRW ; q<img_leftRGB.cols-subRW ; q++){
						int left= supReg.at<int>(p,q,0);				//left arm
						int right= supReg.at<int>(p,q,1);				//right arm
						out.at<double>(p,q,d) = in.at<double>(p,q+right,d)-in.at<double>(p,q-left-1,d);
					}
				}
			}
		break;
		case 'V':
			for(int d=0; d<dispMax-dispMin+1; d++){
				for(int q= subRW ; q<img_leftRGB.cols-subRW ; q++){
					for(int p=subRH ; p<img_leftRGB.rows-subRH ; p++){		
						int up= supReg.at<int>(p,q,2);					//up arm
						int down= supReg.at<int>(p,q,3);				//down arm
						out.at<double>(p,q,d) = in.at<double>(p+down,q,d)-in.at<double>(p-up-1,q,d);
						
					}
				}
			}
		break;
		case 'C':											//copy to final aggregated result volume
			for(int d=0; d<dispMax-dispMin+1; d++){
				for(int p=subRH ; p<img_leftRGB.rows-subRH ; p++){					
					for(int q= subRW ; q<img_leftRGB.cols-subRW ; q++){
						out.at<double>(p,q,d) = in.at<double>(p,q,d);
						
						if(q==subRW)
							left_cost.at<double>(p,q,d)=in.at<double>(p,q,d);
						if (q==img_leftRGB.cols-subRW-1 )
							right_cost.at<double>(p,q,d)=in.at<double>(p,q,d);
						if (p==subRH)
							up_cost.at<double>(p,q,d)=in.at<double>(p,q,d);
						if (p==img_leftRGB.rows-subRH-1)
							down_cost.at<double>(p,q,d)=in.at<double>(p,q,d);
						
						//if( p==243 && q==4)
							//printf("Final: aggcost[%d][%d][%d]= %Lf \t, iter: %d\t\n ", p, q, d, aggr_cost.at<double>(p,q,d), count);
					}
				}
			}
		break;
		default:
			std::cout << "none of the directions"<<std::endl;
		break;
	}
}

/* Calculate color difference */
double imageGpu::colDiffer(cv::Mat in, int x1, int y1, int x2, int y2){
	
	double color = (double) std::max(std::max(abs(in.at<cv::Vec3b>(x1,y1)[0]-in.at<cv::Vec3b>(x2,y2)[0]), abs(in.at<cv::Vec3b>(x1,y1)[1]-in.at<cv::Vec3b>(x2,y2)[1])), 
					abs(in.at<cv::Vec3b>(x1,y1)[2]-in.at<cv::Vec3b>(x2,y2)[2]));
					
	return color;
}

/* Scanline optimization from 4 direction: LRUD */
Mat imageGpu::scanline(double P1, double P2, double lim, Mat& disp, Mat& cost, bool dispR){
	double minLeft=0.0;
	double minRight=0.0;
	double minUp=0.0;
	double minDown=0.0;
	//Left path opt
	for(int p= subRH ; p<img_leftRGB.rows-subRH ; p++){				//Excluding boundaries
		for(int q= subRW+1 ; q<img_leftRGB.cols-subRW ; q++){				
			minLeft=0.0;
			minLeft=MinPathCost(left_cost, p,q-1);
			if(!dispR){
				for(int d=0; d<dispMax-dispMin+1; d++){
					if(q-d-dispMin>subRW){									// > 0 because in calculation of parameters P1 and P2 for Left path optimization, 
																			//the intensity of the the previous pixel on the left is required, which causes an out of boundry error in case of (q-d-dispMin=0)
						left_cost.at<double>(p,q,d) = costOpt(left_cost, p,q,d, minLeft, 'L', P1, P2, lim);
					}	
				}
			} else {
				for(int d=0; d<dispMax-dispMin+1; d++){
					if(q+d+dispMin<img_leftRGB.cols-subRW){									// > 0 because in calculation of parameters P1 and P2 for Left path optimization, 
																			//the intensity of the the previous pixel on the left is required, which causes an out of boundry error in case of (q-d-dispMin=0)
						left_cost.at<double>(p,q,d) = costOpt(left_cost, p,q,d, minLeft, 'L', P1, P2, lim, dispR);
					}	
				}
			}
		}
	}
	
	//Right path opt
	for(int p=subRH ; p<img_leftRGB.rows-subRH ; p++){				//Excluding boundaries
		for(int q=img_leftRGB.cols-subRW-2 ; q>=subRW ; q--){				
			minRight=0.0;
			minRight=MinPathCost(right_cost, p,q+1);
			if(!dispR){
				for(int d=0; d<dispMax-dispMin+1; d++){
					if(q-d-dispMin>subRW-1)
						right_cost.at<double>(p,q,d) = costOpt(right_cost, p,q,d, minRight, 'R', P1, P2, lim);
				}
			} else {
				for(int d=0; d<dispMax-dispMin+1; d++){
					if(q+d+dispMin<img_leftRGB.cols-subRW-1)
						right_cost.at<double>(p,q,d) = costOpt(right_cost, p,q,d, minRight, 'R', P1, P2, lim,dispR);
				}
			}
		}
	}
	
	//Up path opt
	for(int q= subRW; q< img_leftRGB.cols-subRW ; q++){				//Excluding boundaries
		for(int p= subRH+1 ; p<img_leftRGB.rows-subRH ; p++){					
			minUp=0.0;
			minUp=MinPathCost(up_cost, p-1,q);
			if(!dispR){
				for(int d=0; d<dispMax-dispMin+1; d++){
					if(q-d-dispMin>subRW-1)
						up_cost.at<double>(p,q,d) = costOpt(up_cost, p,q,d, minUp, 'U', P1, P2, lim);
				}
			} else {
				for(int d=0; d<dispMax-dispMin+1; d++){
					if(q+d+dispMin<img_leftRGB.cols-subRW)
						up_cost.at<double>(p,q,d) = costOpt(up_cost, p,q,d, minUp, 'U', P1, P2, lim, dispR);
				}					
			}
		}
	}
	
	//Down Path cost
	for(int q= subRW; q< img_leftRGB.cols-subRW ; q++){			//Excluding boundaries
		for(int p= img_leftRGB.rows-subRH-2 ; p>=subRH ; p--){			
			minDown=0.0;
			minDown=MinPathCost(down_cost, p+1,q);
			if(!dispR){
				for(int d=0; d<dispMax-dispMin+1; d++){
					if(q-d-dispMin>subRW-1)
						down_cost.at<double>(p,q,d) = costOpt(down_cost, p,q,d, minDown, 'D', P1, P2, lim);
				}
			} else {
				for(int d=0; d<dispMax-dispMin+1; d++){
					if(q+d+dispMin<img_leftRGB.cols-subRW)
						down_cost.at<double>(p,q,d) = costOpt(down_cost, p,q,d, minDown, 'D', P1, P2, lim, dispR);
				}
			}
		}
	}
	
	finalCost(left_cost,right_cost,down_cost,up_cost, final_cost);
	find_disparity(final_cost, disp, cost);
	return final_cost;
	//subpxEnhance(final_cost,disp);
	
}

/* Minimum path cost of pixel predecessor */
double imageGpu::MinPathCost(cv::Mat in, int p, int q){
	//double min_cost=1.79769e+308;
	double min_cost=in.at<double>(p,q,0);
	for(int d=0; d<dispMax-dispMin+1; d++){
		if(in.at<double>(p,q,d)<=min_cost){
			min_cost = in.at<double>(p,q,d);
		}
		//printf("MinPathCost: cost[%d][%d][%d]: %Lf\t\n", p,q,d, in[p][q][d]);
	}
	return min_cost;
}


/* Take the average of all the path cost */
void imageGpu::finalCost(cv::Mat Lpath, cv::Mat Rpath, cv::Mat Upath, cv::Mat Dpath, cv::Mat outCost){
	int loc_x=0, loc_y=0, loc_d=0;
	double max=0.0;
	double min = 1.79769e+308;
	for(int d=0; d<dispMax-dispMin+1; d++){
		for(int p=subRH ; p<img_leftRGB.rows-subRH ; p++){					
			for(int q= subRW ; q<img_leftRGB.cols-subRW ; q++){
				outCost.at<double>(p,q,d) = (Lpath.at<double>(p,q,d) + Rpath.at<double>(p,q,d) + Upath.at<double>(p,q,d) + Dpath.at<double>(p,q,d))/4.0;
				//printf("final_cost(%d,%d,%d): %f\t, left: %f\t, right: %f\t, up: %f\t, down: %f\t\n", p,q,d, outCost.at<double>(p,q,d), Lpath.at<double>(p,q,d), Rpath.at<double>(p,q,d), Upath.at<double>(p,q,d), \
				Dpath.at<double>(p,q,d));
				
				if (max< outCost.at<double>(p,q,d)){
					max=outCost.at<double>(p,q,d);
					loc_x=q;
					loc_y=p;
					loc_d=d;
				}
				if (outCost.at<double>(p,q,d) < min){
					min = outCost.at<double>(p,q,d);
				}
			}
		}
	}
	std::cout<< "min n max value in the whole cost image: " << min << " , " << max << "  x: " << loc_x<< " , y: " << loc_y << " d: " << loc_d <<std::endl;
}



/* Find the final disparity for each pixel based on WTA method */
void imageGpu::find_disparity(cv::Mat in, cv::Mat& idisp ,cv::Mat& icost){
	for(int p=subRH ; p<img_leftRGB.rows-subRH ; p++){					
		for(int q= subRW ; q<img_leftRGB.cols-subRW ; q++){
			//double tmpcost=1.79769e+308;
			double tmpcost=in.at<double>(p,q,0);
			
			for(int d=0; d<dispMax-dispMin+1; d++){
				if(in.at<double>(p,q,d)<= tmpcost){
					tmpcost=in.at<double>(p,q,d);
					idisp.at<float>(p,q)=(float)d+dispMin;				//Check again later to make sure it's correct..
					icost.at<float>(p,q)=tmpcost;
				}
			}
			
		}
	}
	double minv, maxv;
	cv::Point minL, maxL;
	cv::minMaxLoc(icost, &minv,&maxv, &minL, &maxL);
	std::cout<< "minv: " << (float)minv << " maxv: " << (float)maxv << " maxLx: " << maxL.x << " maxL.y: " << maxL.y << std::endl;
}

void imageGpu::fMatrix(cv::Mat pixflag, cv::Mat dispL, cv::Mat& FM, int N, double param1, double param2 ){
	vector<cv::Point2f> pointsL;
	vector<cv::Point2f> pointsR;
	int n=0;
	for(int p=subRH ; p<img_leftRGB.rows-subRH ; p++){					
		for(int q= subRW ; q<img_leftRGB.cols-subRW ; q++){
			if(n<=N){
				if(pixflag.at<float>(p,q)==0){ 							//reliable pixel
					cv::Point2f p1= Point((float)q, (float)p);
					pointsL.push_back(p1);
					cv::Point2f p2= Point((float)(q-(int)dispL.at<float>(p,q)), (float)p);
					pointsR.push_back(p2);
					n++;
				}
			} else break;
		}
	}

	FM = findFundamentalMat(pointsL, pointsR,FM_RANSAC);
	cout << "fundamental matrix: " << FM << endl;
}


/* Detecting and labeling outliers */
int imageGpu::findOutliers(cv::Mat dispL, cv::Mat dispR,cv::Mat& pixflag, float f, float B){
	int n=0;
	cv::Mat Ql= (Mat_<float>(4,4) << 1,0,0, -(img_leftRGB.cols/2),0,1,0,-(img_leftRGB.rows/2),0,0,0,f,0,0, -1/B, 0);
	    cv::Mat Qr= (Mat_<float>(4,4) << 1,0,0, -(img_leftRGB.cols/2),0,1,0,-(img_leftRGB.rows/2),0,0,0,f,0,0, -1/B, 0);
	cv::Mat T= (Mat_<float>(4,4) << 1,0,0, B,0,1,0,0,0,0,1,0,0,0, 0, 1);
	for(int p=subRH ; p<img_leftRGB.rows-subRH ; p++){					
		for(int q= subRW ; q<img_leftRGB.cols-subRW ; q++){
			//cout<< "dispL: " << dispL.at<float>(p,q) << " dispR: " << dispR.at<float>(p,q-(int)dispL.at<float>(p,q)) << endl;
			if(dispL.at<float>(p,q) != dispR.at<float>(p,q-(int)dispL.at<float>(p,q))){
				//cout<< "dispL: " << dispL.at<float>(p,q) << " dispR: " << dispR.at<float>(p,q-(int)dispL.at<float>(p,q)) << endl;
				n++;
				int occluded = labelOut(Ql, (float)p, (float)q, dispL.at<float>(p,q), dispR.at<float>(p,q-(int)dispL.at<float>(p,q)));
				if(occluded == 1){
					pixflag.at<int>(p,q)=1;				//Occluded
				//cout << dispR.at<float>(p,q-(int)dispL.at<float>(p,q)) << endl;
				}
				else pixflag.at<int>(p,q)=-1;			//Mistmatch
			}
			
		}
	}
	cout<< n << " outlier!" << endl;
	
}

int imageGpu::labelOut(cv::Mat Ql, float pl,float ql, float dl, float dr){
	cv::Mat p1= (Mat_<float>(4,1) << ql,pl,dl,1);
	cv::Mat q1= (Mat_<float>(4,1) << ql-dl, pl, dr, 1);
	cv::Mat p2= (Mat_<float>(4,1) << (ql-dl)+dr, pl, dr, 1);
	cv::Mat res1,res2, res3,res4;
	gemm(p1,Ql, 1, 0, 0, res1, GEMM_1_T + GEMM_2_T);
	//gemm(res1,T, 1, 0, 0, res2, 0);
	//gemm(res2, Qr,1, 0, 0, res3, 0);
	gemm(Ql,p2, 1, 0, 0, res2, 0);
	gemm(res1,res2, 1, 0, 0, res3, 0);
	
	gemm(Ql,p1, 1, 0, 0, res4, 0);
	float mgt1 = (float) sqrt((res2.at<float>(0)*res2.at<float>(0))+ (res2.at<float>(1)*res2.at<float>(1)) 
				+ (res2.at<float>(2)*res2.at<float>(2))+ (res2.at<float>(3)*res2.at<float>(3)));
	float mgt2 = (float) sqrt((res4.at<float>(0)*res4.at<float>(0))+ (res4.at<float>(1)*res4.at<float>(1)) 
				+ (res4.at<float>(2)*res4.at<float>(2))+ (res4.at<float>(3)*res4.at<float>(3)));
	
	float RE1 = res3.at<float>(0);
	float RE2 = mgt1 * mgt2;
	//cout << "res4: " << res4 << " res3: " << res3 << " mgt1: " <<mgt1 << " mgt2: " << mgt2 << " multi: " << RE2 << " " << RE1<< endl;
	//cout << " abs: " << abs(RE1-RE2) << endl;
	if(abs(RE1-RE2) < 2){											//Threshold? (Float calculation difference?)
		//cout << " Mismatch" << endl;
		return 0;
	}
	else {
		//cout << " Occluded!" << endl;
		return 1;
	}
	
	//cv::Mat zero = Mat::zeros(1,1,CV_32F);
	//cv::Mat diff = res4 != zero;
	// Equal if no elements disagree
	//bool equal = cv::countNonZero(diff) == 0;
	/*if(equal){
		cout << res4 << " Mismatch" << endl;
	}*/
	//else cout << "Occluded" << endl;
}

void imageGpu::interpolate(cv::Mat img, cv::Mat& disp, cv::Mat pixflag){
	
	for(int p=subRH ; p<img_leftRGB.rows-subRH ; p++){					
		for(int q= subRW ; q<img_leftRGB.cols-subRW ; q++){
			
			if(pixflag.at<int>(p,q)==1){ 				//Occluded
				float min_disp = 10000000;
				float min1, min2;
				for(int j=-2; j<3; j++){
					
					min1 = std::min(((p-2 < subRH || q+j > img_leftRGB.cols-subRW-1 || q+j < subRW || pixflag.at<int>(p-2,q+j)!=0) ? 20000000 : disp.at<float>(p-2,q+j)), 
									((p+2 > img_leftRGB.rows-subRH-1 || q+j > img_leftRGB.cols-subRW-1 || q+j < subRW || pixflag.at<int>(p+2,q+j)!=0) ? 20000000 : disp.at<float>(p+2,q+j)));
					if(min1 < min_disp){
						min_disp = min1;
					}
				}
				for (int i=-1; i<2; i++){
					min1 = std::min(((p+i < subRH || p+i > img_leftRGB.rows-subRH-1 || q-2 < subRW || pixflag.at<int>(p+i,q-2)!=0) ? 20000000 : disp.at<float>(p+i,q-2)),
									((p+i < subRH || p+i > img_leftRGB.rows-subRH-1 || q+2 > img_leftRGB.cols-subRW-1 || pixflag.at<int>(p+i,q+2)!=0) ? 20000000 : disp.at<float>(p+i, q+2)));
					if(min1 < min_disp){
						min_disp = min1;
					}
				}
				if(min_disp == 10000000 || min_disp == 20000000) ; //cout << "Very large disp! " << endl;
				else disp.at<float>(p,q) = min_disp;							//lowest disparity from 16 nearest reliable pixels
			}
			else if(pixflag.at<int>(p,q)== -1){ 				//Mismatch
				int x,y, tempx, tempy;
				Mat gimg;
				int min_diff = 10000000;
				int temp_diff;
				int diff1, diff2;
				unsigned char I2, I3;
				cvtColor(img, gimg, CV_BGR2GRAY);
				unsigned char I1 = gimg.at<uchar>(p,q);
				for(int j=-2; j<3; j++){
					I2 =  gimg.at<uchar>(p-2,q+j);
					I3 =  gimg.at<uchar>(p+2,q+j);
					diff1 = (p-2 < subRH || q+j < subRW || q+j > img_leftRGB.cols-subRW-1 || pixflag.at<int>(p-2,q+j)!=0) ? 20000000 : abs((int)I1 - I2);
					diff2 = (p+2 > img_leftRGB.rows-subRH-1 || q+j < subRW || q+j > img_leftRGB.cols-subRW-1 || pixflag.at<int>(p+2,q+j)!=0) ? 20000000 : abs((int)I1 - I3);
					if(diff1 < diff2){
						temp_diff = diff1;
						tempx = q+j;
						tempy = p-2;
					} else {
						temp_diff = diff2;
						tempx = q+j;
						tempy = p+2;
					}
					if(temp_diff < min_diff ){
						min_diff = temp_diff;
						x = tempx;
						y = tempy;
					}
				}
				for (int i=-1; i<2; i++){
					I2 = gimg.at<uchar>(p+i,q-2);
					I3 = gimg.at<uchar>(p+i,q+2);
					diff1 = (q-2 < subRW || p+i < subRH || p+i > img_leftRGB.rows-subRH-1 || pixflag.at<int>(p+i,q-2)!=0) ? 20000000 : abs((int)I1 - I2);
					diff2 = (q+2 > img_leftRGB.cols-subRW-1 || p+i < subRH || p+i > img_leftRGB.rows-subRH-1 || pixflag.at<int>(p+i,q+2)!=0) ? 20000000 : abs((int)I1 - I3);
					if(diff1 < diff2){
						temp_diff = diff1;
						tempx = q-2;
						tempy = p+i;
					} else {
						temp_diff = diff2;
						tempx = q+2;
						tempy = p+i;
					}
					if(temp_diff < min_diff){
						min_diff = temp_diff;
						x = tempx;
						y = tempy;
					}
				}
				if(min_diff == 20000000 || min_diff == 10000000) ;//cout << "Very large intensity difference! " << endl;
				else disp.at<float>(p,q) = disp.at<float>(y,x);
			}
			
		}
	}
}

void imageGpu::border(cv::Mat disp, cv::Mat& grad){
	//cv::Mat Gx = (Mat_<float>(3,3) << 1,0,-1, 2,0,-2, 1,0,-1);
	//cv::Mat Gy = (Mat_<float>(3,3) << 1,2,1, 0,0,0, -1,-2,-1);
	int scale = 1;
	int delta = 0;
	int ddepth = CV_16S;
	Mat grad_x, grad_y;
	Mat abs_grad_x, abs_grad_y;

	/// Gradient X
	Scharr( disp, grad_x, ddepth, 1, 0, scale, delta, BORDER_DEFAULT );
	//Sobel( disp, grad_x, ddepth, 1, 0, 3, scale, delta, BORDER_DEFAULT );
	convertScaleAbs( grad_x, abs_grad_x );

	/// Gradient Y
	Scharr( disp, grad_y, ddepth, 0, 1, scale, delta, BORDER_DEFAULT );
	//Sobel( disp, grad_y, ddepth, 0, 1, 3, scale, delta, BORDER_DEFAULT );
	convertScaleAbs( grad_y, abs_grad_y );

	/// Total Gradient (approximate)
	addWeighted( abs_grad_x, 1, abs_grad_y, 1, 0, grad );
	
}

void imageGpu::discAdjust(cv::Mat& disp, cv::Mat fcost, cv::Mat mask){
	//std::list<Point> pt;
	int n=0;
	for(int p=subRH+1 ; p<img_leftRGB.rows-subRH-1 ; p++){					
		for(int q= subRW+1 ; q<img_leftRGB.cols-subRW-1 ; q++){
			n=0;
			if(mask.at<int>(p,q) != 0){
				
				if(mask.at<int>(p,q-1) ==0){
					n++;
					//cout << " Left Not On Edge " << endl;
					//Point p1= Point(q-1,p);
					//pt.push_back(p1);
				}
				if(mask.at<int>(p,q+1) == 0){
					n++;
				}
				//cout << "size: " << pt.size() << endl;
				if(n==2){
					cout << " 2 points found!" << endl;
				
					if(fcost.at<double>(p,q-1,(int)disp.at<float>(p,q-1)) < fcost.at<double>(p,q,(int)disp.at<float>(p,q))){
						disp.at<float>(p,q) = disp.at<float>(p,q-1);
					}
					if(fcost.at<double>(p,q+1,(int)disp.at<float>(p,q+1)) < fcost.at<double>(p,q,(int)disp.at<float>(p,q))){
						disp.at<float>(p,q) = disp.at<float>(p,q+1);
					}
					
				}
				
			}
		}
	}
	
}


/* Iterative region voting */
void imageGpu::regionVoting(cv::Mat& dispL, cv::Mat& pixflag, int TS, double TH,int iter){
	int n=0;
	Mat hist(dispMax-dispMin+1,1,CV_32SC1, Scalar::all(0));
	int reliables=0;				//Number of reliable pixels
	
	while(n<iter){
		for(int p=subRH ; p<img_leftRGB.rows-subRH ; p++){					
			for(int q= subRW ; q<img_leftRGB.cols-subRW ; q++){
				if(pixflag.at<int>(p,q)!=0){
					//cout << "p,q " << p << " , " << q << endl;
						reliables=0;
						hist=Scalar::all(0);
						int up= supReg.at<int>(p,q,2);
						int down= supReg.at<int>(p,q,3);
						//cout << "up: " << up << " Down: " << down << endl;
						for(int y=p-up;y<=p+down; y++){
							int left= supReg.at<int>(y,q,0);
							int right= supReg.at<int>(y,q,1);
							//cout <<  y << " , " << q <<  " left: " << left << " right: " << right << endl;
							for(int x=q-left; x<=q+right; x++){
								//if(x!=q || y!=p){							//Don't involve the pixel of interest in calculation -> NOT necessary the next 'if' excludes it by itself
									if(pixflag.at<int>(y,x)==0){				//Reliable pixel
										reliables++;
										hist.at<int>((int)dispL.at<float>(y,x)) +=1;
									}
									//cout << "reliable: " << reliables << endl;
								//}
							}
						}
						int newd= mostVote(hist);
						//cout << "hist: " << hist << '\t' << " most vote: " << newd << endl;
						if(reliables > TS && ( ((double)hist.at<int>(newd,0)/reliables) > TH)) {
							//cout <<  p << " , " << q << " original disp: " << dispL.at<float>(p,q) << "  updated disp: " << (float) newd +  dispMin<< endl;
							dispL.at<float>(p,q) = (float) newd + dispMin;
							pixflag.at<int>(p,q) = 0;
						}
				}
			} 
		}
		n++;
	}
}

int imageGpu::mostVote(cv::Mat hist){
	int votes = hist.at<int>(0,0);
	int index=0;
	for (int i=0; i<hist.rows; i++){
		if(hist.at<int>(i,0)>=votes){
			votes = hist.at<int>(i,0);
			index=i;
		}
	}
	return index;
}


/* Calculating ech path cost to a pixel of interest */
double imageGpu::costOpt(cv::Mat in, int p, int q, int d, double preMin, char dir, double param1, double param2, double threshold,bool dispR){
	double cost=0.0;
	std::pair<double,double> P;
	if(!dispR){
		switch (dir){
			case 'L':
				P = calc_param(p,q,p,q-1, p,q-d-dispMin,p,q-d-dispMin-1, threshold, param1, param2);
				if(!dispValid(d-1) && dispValid(d+1)){
					cost=aggr_cost.at<double>(p,q,d)+minimum(in.at<double>(p,q-1,d), in.at<double>(p,q-1,d+1)+P.first,preMin+P.second)- preMin;
				}
				else if (dispValid(d-1) && !(dispValid(d+1))){
					cost=aggr_cost.at<double>(p,q,d)+minimum(in.at<double>(p,q-1,d), in.at<double>(p,q-1,d-1)+P.first,preMin+P.second)- preMin;
				}
				else {
					cost=aggr_cost.at<double>(p,q,d)+minimum(in.at<double>(p,q-1,d), in.at<double>(p,q-1,d-1)+P.first, in.at<double>(p,q-1,d+1)+P.first,preMin+P.second) - preMin;
				}
				/*if(p==4 && q==7 && d==5){
					printf("CostOpt: aggr_cost[%d][%d][%d]= %Lf \t , minLeft:%Lf\t\n , pre_d: %Lf\t, pre_d-1: %Lf\t, param1: %f\t, param2: %f\t\n", p, q, d, 
					aggr_cost[p][q][d], preMin, in[p][q-1][d], in[p][q-1][d-1]+P.first,P.first, P.second);
				}*/
			break;
			
			case 'R':
				P = calc_param(p,q,p,q+1, p,q-d-dispMin,p,q-d-dispMin+1, threshold, param1, param2);
				if(!dispValid(d-1) && dispValid(d+1)){
					cost=aggr_cost.at<double>(p,q,d)+minimum(in.at<double>(p,q+1,d), in.at<double>(p,q+1,d+1)+P.first,preMin+P.second)- preMin;
				}
				else if (dispValid(d-1) && !(dispValid(d+1))){
					cost=aggr_cost.at<double>(p,q,d)+minimum(in.at<double>(p,q+1,d), in.at<double>(p,q+1,d-1)+P.first,preMin+P.second)- preMin;
				}
				else {
					cost=aggr_cost.at<double>(p,q,d)+minimum(in.at<double>(p,q+1,d), in.at<double>(p,q+1,d-1)+P.first, in.at<double>(p,q+1,d+1)+P.first,preMin+P.second) - preMin;
				}
			break;
				
			case 'U':
			
				P = calc_param(p,q,p-1,q, p,q-d-dispMin,p-1,q-d-dispMin, threshold, param1, param2);
				if(!dispValid(d-1) && dispValid(d+1)){
					cost=aggr_cost.at<double>(p,q,d)+minimum(in.at<double>(p-1,q,d), in.at<double>(p-1,q,d+1)+P.first,preMin+P.second)- preMin;
				}
				else if (dispValid(d-1) && !(dispValid(d+1))){
					cost=aggr_cost.at<double>(p,q,d)+minimum(in.at<double>(p-1,q,d), in.at<double>(p-1,q,d-1)+P.first,preMin+P.second)- preMin;
				}
				else {
					cost=aggr_cost.at<double>(p,q,d)+minimum(in.at<double>(p-1,q,d), in.at<double>(p-1,q,d-1)+P.first, in.at<double>(p-1,q,d+1)+P.first,preMin+P.second) - preMin;
				}
			break;
			
			case 'D':
				P = calc_param(p,q,p+1,q, p,q-d-dispMin,p+1,q-d-dispMin, threshold, param1, param2);
				if(!dispValid(d-1) && dispValid(d+1)){
					cost=aggr_cost.at<double>(p,q,d)+minimum(in.at<double>(p+1,q,d), in.at<double>(p+1,q,d+1)+P.first,preMin+P.second)- preMin;
				}
				else if (dispValid(d-1) && !(dispValid(d+1))){
					cost=aggr_cost.at<double>(p,q,d)+minimum(in.at<double>(p+1,q,d), in.at<double>(p+1,q,d-1)+P.first,preMin+P.second)- preMin;
				}
				else {
					cost=aggr_cost.at<double>(p,q,d)+minimum(in.at<double>(p+1,q,d), in.at<double>(p+1,q,d-1)+P.first, in.at<double>(p+1,q,d+1)+P.first,preMin+P.second) - preMin;
				}
			break;
			
			default:
				std::cout << "no direction specified" <<std::endl;
				break;
		}
	} else {					//Right disparity
		switch (dir){
			case 'L':
				P = calc_param(p,q+d+dispMin,p,q+d+dispMin-1, p,q,p,q-1, threshold, param1, param2);
				if(!dispValid(d-1) && dispValid(d+1)){
					cost=aggr_cost.at<double>(p,q,d)+minimum(in.at<double>(p,q-1,d), in.at<double>(p,q-1,d+1)+P.first,preMin+P.second)- preMin;
				}
				else if (dispValid(d-1) && !(dispValid(d+1))){
					cost=aggr_cost.at<double>(p,q,d)+minimum(in.at<double>(p,q-1,d), in.at<double>(p,q-1,d-1)+P.first,preMin+P.second)- preMin;
				}
				else {
					cost=aggr_cost.at<double>(p,q,d)+minimum(in.at<double>(p,q-1,d), in.at<double>(p,q-1,d-1)+P.first, in.at<double>(p,q-1,d+1)+P.first,preMin+P.second) - preMin;
				}
				/*if(p==4 && q==7 && d==5){
					printf("CostOpt: aggr_cost[%d][%d][%d]= %Lf \t , minLeft:%Lf\t\n , pre_d: %Lf\t, pre_d-1: %Lf\t, param1: %f\t, param2: %f\t\n", p, q, d, 
					aggr_cost[p][q][d], preMin, in[p][q-1][d], in[p][q-1][d-1]+P.first,P.first, P.second);
				}*/
			break;
			
			case 'R':
				P = calc_param(p,q+d+dispMin,p,q+d+dispMin+1, p,q,p,q+1, threshold, param1, param2);
				if(!dispValid(d-1) && dispValid(d+1)){
					cost=aggr_cost.at<double>(p,q,d)+minimum(in.at<double>(p,q+1,d), in.at<double>(p,q+1,d+1)+P.first,preMin+P.second)- preMin;
				}
				else if (dispValid(d-1) && !(dispValid(d+1))){
					cost=aggr_cost.at<double>(p,q,d)+minimum(in.at<double>(p,q+1,d), in.at<double>(p,q+1,d-1)+P.first,preMin+P.second)- preMin;
				}
				else {
					cost=aggr_cost.at<double>(p,q,d)+minimum(in.at<double>(p,q+1,d), in.at<double>(p,q+1,d-1)+P.first, in.at<double>(p,q+1,d+1)+P.first,preMin+P.second) - preMin;
				}
			break;
				
			case 'U':
			
				P = calc_param(p,q+d+dispMin,p-1,q+d+dispMin, p,q,p-1,q, threshold, param1, param2);
				if(!dispValid(d-1) && dispValid(d+1)){
					cost=aggr_cost.at<double>(p,q,d)+minimum(in.at<double>(p-1,q,d), in.at<double>(p-1,q,d+1)+P.first,preMin+P.second)- preMin;
				}
				else if (dispValid(d-1) && !(dispValid(d+1))){
					cost=aggr_cost.at<double>(p,q,d)+minimum(in.at<double>(p-1,q,d), in.at<double>(p-1,q,d-1)+P.first,preMin+P.second)- preMin;
				}
				else {
					cost=aggr_cost.at<double>(p,q,d)+minimum(in.at<double>(p-1,q,d), in.at<double>(p-1,q,d-1)+P.first, in.at<double>(p-1,q,d+1)+P.first,preMin+P.second) - preMin;
				}
			break;
			
			case 'D':
				P = calc_param(p,q+d+dispMin,p+1,q+d+dispMin, p,q,p+1,q, threshold, param1, param2);
				if(!dispValid(d-1) && dispValid(d+1)){
					cost=aggr_cost.at<double>(p,q,d)+minimum(in.at<double>(p+1,q,d), in.at<double>(p+1,q,d+1)+P.first,preMin+P.second)- preMin;
				}
				else if (dispValid(d-1) && !(dispValid(d+1))){
					cost=aggr_cost.at<double>(p,q,d)+minimum(in.at<double>(p+1,q,d), in.at<double>(p+1,q,d-1)+P.first,preMin+P.second)- preMin;
				}
				else {
					cost=aggr_cost.at<double>(p,q,d)+minimum(in.at<double>(p+1,q,d), in.at<double>(p+1,q,d-1)+P.first, in.at<double>(p+1,q,d+1)+P.first,preMin+P.second) - preMin;
				}
			break;
			
			default:
				std::cout << "no direction specified" <<std::endl;
				break;
		}
	}
	return cost;
}

bool imageGpu::dispValid(int d){
	return (d < 0 || d >dispMax-dispMin) ? false:true;
}

double imageGpu::minimum(double a, double b, double c, double d){
	if(d==-100){
		return a < b ? (a < c ? a : c) : (b < c ? b : c);
	}
	else {
		double tmp1 = a < b ? a : b;
		double tmp2 = c < d ? c : d;
		return tmp1 < tmp2 ? tmp1 : tmp2;
	}
}


/* Calculating some of the constants for the algorithm */
std::pair<double,double> imageGpu::calc_param(int x1, int y1, int x2, int y2, int x3, int y3, 
									int x4, int y4, double thr, double const1, 
									double const2){
	
	double param1=0, param2=0;
	std::pair <double,double> p1;
	double leftdiff= colDiffer(img_leftRGB, x1,y1,x2,y2);
	double rightdiff= colDiffer(img_rightRGB, x3,y3,x4,y4);
	if(leftdiff<thr && rightdiff<thr)
	{
		param1=const1;
		param2=const2;
	}
	else if((leftdiff<thr && rightdiff>thr) || (leftdiff>thr && rightdiff<thr))
	{
		param1=const1/4;
		param2=const2/4;
	}
	else  					///FIXME: Here also includes when the values are equal, but in the paper it's just when they are bigger than threshold. Test it to see...
	{
		param1=const1/10;		
		param2=const1/10;
	}
	
	p1 = std::make_pair(param1, param2);
	return p1;
}

double imageGpu::findMax(cv::Mat in){
	double max=0.0;
	int x,y;
	for(int d=0; d<dispMax-dispMin+1; d++){
		for(int p=subRH ; p<img_leftRGB.rows-subRH ; p++){					
			for(int q= subRW ; q<img_leftRGB.cols-subRW ; q++){
				if(max<in.at<double>(p,q,d)){
					max=in.at<double>(p,q,d);
					x=q;
					y=p;
				}
			}
		}
	}
	//std::cout<< "x: " << x << " y: " << y << std::endl;
	return max;
}


void imageGpu::subpxEnhance(cv::Mat fcost, cv::Mat& idisp){
	for(int p=subRH ; p<img_leftRGB.rows-subRH ; p++){					
		for(int q= subRW ; q<img_leftRGB.cols-subRW ; q++){
				int d = idisp.at<float>(p,q)-dispMin;
				//std::cout << "float d: " << idisp.at<float>(p,q)<< " int d: " << d << std::endl;
				if(d>0 && d < dispMax-dispMin){
					double val = (fcost.at<double>(p,q,d+1) - fcost.at<double>(p,q,d-1))/(2*(fcost.at<double>(p,q,d+1)+
					fcost.at<double>(p,q,d-1) - 2*fcost.at<double>(p,q,d)));
				//	std::cout << "old disp: " << idisp.at<float>(p,q) <<std::endl;
					idisp.at<float>(p,q) = idisp.at<float>(p,q) - (float)val;
				//	std::cout << "new disp: " << idisp.at<float>(p,q) <<std::endl;
				}
			}
	}
	cv::medianBlur(idisp, idisp, 3);
	//cv::GaussianBlur(idisp, idisp, cv::Size(3,3), 3,3, cv::BORDER_DEFAULT);
}
